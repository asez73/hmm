#include "hip/hip_runtime.h"
/*
**      Edited by: Philip Yang, Juncheng Chen
**      Date:   2 Dec 2010
**      File:   viterbi_kernel.cu
**      Purpose: Viterbi algorithm for computing the maximum likelihood
**		state sequence and probablity of observing a sequence
**		given the model. The new code is supposed to run on
**              nVidia's CUDA capable hardware
**      Organization: University of Maryland
**
*/

#include "hmm.h"


//#define real float

/// this kernel deals with 
/**
 *\brief Viterbi O(N^2) inner loop
 *
 *\param Symbol the T-th observed symbol
 *\param delta_prev previous delta vector
 *\param delta_curr current delta vector to be updated
 *\param A the state transition matrix, in column major order
 *\param B the state emission matrix, in column major order
 *\param N number of states
 */
__global__ void ViterbiKernel(int Symbol, real* delta_prev, real* delta_curr, int* psi_curr, real* A, real *B, size_t N)
{
  /// compute the distribution of threads
  /// using 1D layout  

  int pb_stride = blockDim.x * gridDim.x;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //int stride = num_threads / size;

  // each thread to be fixed at the tid-th column
  for ( int j=idx; j<N; j += pb_stride )
    { 
      real maxval = 0.0;
      real maxvalidx = 1;
      for ( int i=0; i<N; ++i )
	{
	  real tmp = delta_prev[i] * A[j*N + i];
	  if ( maxval < tmp )
	    {
	       maxval = tmp;
	       /// Waring: the index must be incremented
	       maxvalidx = i + 1;
	    }
	}
      delta_curr[j] = maxval * B[ (Symbol-1)*N + j ];
      psi_curr[j] = maxvalidx;      
    }  
}



/// this kernel deals with 
/**
 *\brief Viterbi O(N^2) inner loop
 *
 * Compared with the basic version, this new kernel
 * makes each block compute one column
 *
 *\param Symbol the T-th observed symbol
 *\param delta_prev previous delta vector
 *\param delta_curr current delta vector to be updated
 *\param A the state transition matrix, in column major order
 *\param B the state emission matrix, in column major order
 *\param N number of states
 */
__global__ void ViterbiKernelv1(int Symbol, real* delta_prev, real* delta_curr, int* psi_curr, real* A, real *B, size_t N)
{
  /// compute the distribution of threads
  /// using 1D layout  

  int pb_stride = gridDim.x;
  //int stride = num_threads / size;

  const int num_threads = 2;
  __shared__ real maxvalist[num_threads];
  __shared__ int maxvalidx_list[num_threads];

  // each thread to be fixed at the tid-th column
  register real maxval = 0.0;
  register real maxvalidx = 1;
    
  for ( int j=blockIdx.x; j<N; j += pb_stride )
    { 
      for ( int i=threadIdx.x; i<N; i += blockDim.x )
	{
	  real tmp = delta_prev[i] * A[j*N + i];
	  if ( maxval < tmp )
	    {
	       maxval = tmp;
	       /// Waring: the index must be incremented
	       maxvalidx = i + 1;
	    }
	}
      maxvalist[threadIdx.x] = maxval;
      maxvalidx_list[threadIdx.x] = maxvalidx;

      /// block level barrier
      __syncthreads();
      
      if ( threadIdx.x == 0 )
	{
	  maxval = maxvalist[0];
	  maxvalidx = maxvalidx_list[0];
	  for ( int t=1; t<num_threads; ++t )
	    {
	      if ( maxval < maxvalist[t] )
		{
		  maxval = maxvalist[t];
		  maxvalidx = maxvalidx_list[t];
		}
	    }
	  
	  delta_curr[j] = maxval * B[ (Symbol-1)*N + j ];
	  psi_curr[j] = maxvalidx;      
	}
    }  
}

