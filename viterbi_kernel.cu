#include "hip/hip_runtime.h"
/*
**      Edited by: Philip Yang, Juncheng Chen
**      Date:   2 Dec 2010
**      File:   viterbi_kernel.cu
**      Purpose: Viterbi algorithm for computing the maximum likelihood
**		state sequence and probablity of observing a sequence
**		given the model. The new code is supposed to run on
**              nVidia's CUDA capable hardware
**      Organization: University of Maryland
**
*/

#include "hmm.h"


//#define real float

/// this kernel deals with 
/**
 *\brief Viterbi O(N^2) inner loop
 *
 *\param Symbol the T-th observed symbol
 *\param delta_prev previous delta vector
 *\param delta_curr current delta vector to be updated
 *\param A the state transition matrix, in column major order
 *\param B the state emission matrix, in column major order
 *\param N number of states
 */
__global__ void ViterbiKernel(int Symbol, real* delta_prev, real* delta_curr, int* psi_curr, real* A, real *B, size_t N)
{
  /// compute the distribution of threads
  /// using 1D layout  

  int pb_stride = blockDim.x * gridDim.x;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //int stride = num_threads / size;

  // each thread to be fixed at the tid-th column
  for ( int j=idx; j<N; j += pb_stride )
    { 
      real maxval = 0.0;
      real maxvalidx = 1;
      for ( int i=0; i<N; ++i )
	{
	  real tmp = delta_prev[i] * A[j*N + i];
	  if ( maxval < tmp )
	    {
	      maxval = tmp;
	      /// Waring: the index must be incremented
	      maxvalidx = i + 1;
	    }
	}
      delta_curr[j] = maxval * B[ (Symbol-1)*N + j ];
      psi_curr[j] = maxvalidx;      
    }  
}



/// this kernel deals with 
/**
 *\brief Viterbi O(N^2) inner loop
 *
 * Compared with the basic version, this new kernel
 * makes each block compute one column
 *
 *\param Symbol the T-th observed symbol
 *\param delta_prev previous delta vector
 *\param delta_curr current delta vector to be updated
 *\param A the state transition matrix, in column major order
 *\param B the state emission matrix, in column major order
 *\param N number of states
 */
__global__ void ViterbiKernelv1(int Symbol, real* delta_prev, real* delta_curr, int* psi_curr, real* A, real *B, size_t N)
{
  /// compute the distribution of threads
  /// using 1D layout  

  //int pb_stride = gridDim.x;
  //int stride = num_threads / size;

  /// note that this kernel only takes 32 threads 
  const int num_threads = 32;
  __shared__ real maxvalist[num_threads];
  __shared__ int maxvalidx_list[num_threads];
    
  for ( int j=blockIdx.x; j<N; j += gridDim.x )
    {
      /// each thread to be fixed at the tid-th column
      register real maxval = 0.0;
      register real maxvalidx = 1;
  
      for ( int i=threadIdx.x; i<N; i += blockDim.x )
  	{
  	  real tmp = delta_prev[i] * A[j*N + i];
  	  if ( maxval < tmp )
  	    {
	      maxval = tmp;
	      /// Waring: the index must be incremented
	      maxvalidx = i + 1;
  	    }
  	}
      
      maxvalist[threadIdx.x] = maxval;
      maxvalidx_list[threadIdx.x] = maxvalidx;
      __threadfence_block();
      //__syncthreads();
      
      /// we should consider using a better method
      if ( threadIdx.x == 0 )
      	{
      	  maxval = maxvalist[0];
      	  maxvalidx = maxvalidx_list[0];
      	  for ( int t=1; t<num_threads; ++t )
      	    {
      	      if ( maxval < maxvalist[t] )
      		{
      		  maxval = maxvalist[t];
      		  maxvalidx = maxvalidx_list[t];
      		}
      	    }
	  
      	  delta_curr[j] = maxval * B[ (Symbol-1)*N + j ];
      	  psi_curr[j] = maxvalidx;
      	}
      __syncthreads();
    }  
}




/// this kernel deals with 
/**
 *\brief Viterbi O(N^2) inner loop
 *
 * Compared with the basic version, this new kernel
 * makes each block compute one column
 *
 *\param Symbol the T-th observed symbol
 *\param delta_prev previous delta vector
 *\param delta_curr current delta vector to be updated
 *\param A the state transition matrix, in column major order
 *\param B the state emission matrix, in column major order
 *\param N number of states
 */
__global__ void ViterbiLogKernelv1(int Symbol, real* delta_prev, real* delta_curr, int* psi_curr, real* A, real *B, size_t N)
{
  /// compute the distribution of threads
  /// using 1D layout  

  //int pb_stride = gridDim.x;
  //int stride = num_threads / size;

  /// note that this kernel only takes 32 threads 
  const int num_threads = 32;
  __shared__ real maxvalist[num_threads];
  __shared__ int maxvalidx_list[num_threads];
    
  for ( int j=blockIdx.x; j<N; j += gridDim.x )
    {
      /// each thread to be fixed at the tid-th column
      register real maxval = 0.0;
      register real maxvalidx = 1;
  
      for ( int i=threadIdx.x; i<N; i += blockDim.x )
  	{
  	  real tmp = delta_prev[i] + log(A[j*N + i]);
  	  if ( maxval < tmp )
  	    {
	      maxval = tmp;
	      /// Waring: the index must be incremented
	      maxvalidx = i + 1;
  	    }
  	}
      
      maxvalist[threadIdx.x] = maxval;
      maxvalidx_list[threadIdx.x] = maxvalidx;
      __threadfence_block();
      //__syncthreads();
      
      /// we should consider using a better method
      if ( threadIdx.x == 0 )
      	{
      	  maxval = maxvalist[0];
      	  maxvalidx = maxvalidx_list[0];
      	  for ( int t=1; t<num_threads; ++t )
      	    {
      	      if ( maxval < maxvalist[t] )
      		{
      		  maxval = maxvalist[t];
      		  maxvalidx = maxvalidx_list[t];
      		}
      	    }
	  
      	  delta_curr[j] = maxval + log(B[ (Symbol-1)*N + j ]);
      	  psi_curr[j] = maxvalidx;
      	}
      __syncthreads();
    }  
}




__device__ unsigned int count = 0;
__shared__ bool isLastBlockDone;


__device__ void syncAllThreads()
{
  __syncthreads();
  if (threadIdx.x == 0)
    atomicInc(&count, gridDim.x-1);
  volatile unsigned int* counter = &count;
    do
      {
      } while (*counter > 0);
}

/**
 *\brief Viterbi O(TN^2) main loop
 *
 * Compared with the basic version, this new kernel
 * makes each block compute one column
 *
 *\param query the query observation sequence
 *\param delta_prev previous delta vector
 *\param delta_curr current delta vector to be updated
 *\param A the state transition matrix, in column major order
 *\param B the state emission matrix, in column major order
 *\param N number of states
 */
__global__ void ViterbiKernelv2(int* query, int T,
				real* delta, int* psi, 
				real* A, real *B, size_t N)
{
  /// compute the distribution of threads
  /// using 1D layout  

  //int pb_stride = gridDim.x;
  //int stride = num_threads / size;

  /// note that this kernel only takes 64 threads 
  const int num_threads = 32;
  __shared__ real maxvalist[num_threads];
  __shared__ int maxvalidx_list[num_threads];

  
  for ( int t=1; t<T; ++t )
    { 
      for ( int j = blockIdx.x; j<N; j += gridDim.x )
	{
	  /// each thread to be fixed at the tid-th column
	  register real maxval = 0.0;
	  register real maxvalidx = 1;

	  for ( int i=threadIdx.x; i<N; i += blockDim.x )
	    {
	      real tmp = delta[(t-1)*N + i] * A[j*N + i];
	      if ( maxval < tmp )
		{
		  maxval = tmp;
		  /// Waring: the index must be incremented
		  maxvalidx = i + 1;
		}
	    }
	  
	  maxvalist[threadIdx.x] = maxval;
	  maxvalidx_list[threadIdx.x] = maxvalidx;
	  __threadfence_block();
	  __syncthreads();
	  
      
	  /// update the last one here
	  if ( threadIdx.x == 0 )
	    {
	      maxval = maxvalist[0];
	      maxvalidx = maxvalidx_list[0];
	      for ( int btid=1; btid<num_threads; ++btid )
		{
		  if ( maxval < maxvalist[btid] )
		    {
		      maxval = maxvalist[btid];
		      maxvalidx = maxvalidx_list[btid];
		    }
		}
	  
	      delta[t*N + j] = maxval * B[(query[t] - 1)*N + j];
	      psi[t*N + j] = maxvalidx;      

	      __threadfence();
	    }
	  __syncthreads();
	}

      syncAllThreads();
      
    }  
}

